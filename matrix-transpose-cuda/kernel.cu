#include "hip/hip_runtime.h"


#include <stdio.h>


#include <stdio.h>
#define M 16 //row
#define N 16
#define THREAD_PER_BLOCK_X 2;
#define THREAD_PER_BLOCK_Y 2; 

__global__ void transposeMatrix(int *a, int *c)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int column = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * M + column;
	int indexT = column * N + row;
	c[indexT] = a[index];
}

int main()
{
	int a[M][N], c[N][M];
	int *d_a, *d_c;
	int size = sizeof(int) * N * M;
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_c, size);

	//init matrix
	int i, j;
	for (i = 0; i < M; i++) {
		for (j = 0; j < N; j++) {
			a[i][j] = i;
			printf("%d ", a[i][j]);
		}
		printf("\n");
	}
	printf("*********\n");
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	dim3 grid, block;
	grid.x = M / THREAD_PER_BLOCK_X;
	grid.y = N / THREAD_PER_BLOCK_Y;

	block.x = THREAD_PER_BLOCK_X;
	block.y = THREAD_PER_BLOCK_Y;

	transposeMatrix<<<grid, block>>>(d_a, d_c);
	hipDeviceSynchronize(); //is it necessary?

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	//print result
	int m, n;
	for (m = 0; m < M; m++) {
		for (n = 0; n < N; n++) {
			printf("%d ", c[m][n]);
		}
		printf("\n");
	}

	hipFree(d_a);
	hipFree(d_c);

	return 0;
}